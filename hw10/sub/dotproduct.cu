
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TOTAL_THREADS       172032
#define THREADS_PER_BLOCK   128
#define TOTAL_BLOCKS        (TOTAL_THREADS / THREADS_PER_BLOCK) // 1344

#define VECTOR_LENGTH       (5 * TOTAL_THREADS)

__global__ void dotProduct(double *d_c, double *d_a, double *d_b, int length, int valsPerThread)
{
    // declare a buffer in shared memory to hold the partial reductions from each thread
    // in a block. You will need to use a constant value to declare this, so use the
    // number of threads/block that you have computed.
    __shared__ double partial[THREADS_PER_BLOCK];

    // declare a thread local/automatic variable (we'll call it c) in a register to hold
    // the results for each thread in the loop below.
    double c = 0;

    // compute the local dot product for each thread's values
    // each thread will do a multiply and summation across valsPerThread elements
    // of the d_a and d_b vectors. All threads in a block should access adjacent
    // elements. I would suggest all threads on the device accessing a block of
    // data, and then moving on to the next block, and doing this a total of
    // valsPerThread times.
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = 0; i < valsPerThread; i++) {
        c += d_a[idx] * d_b[idx];
        idx += stride;
    }

    // store c into the proper thred position of the shared memory buffer declared
    // above.
    partial[threadIdx.x] = c;

    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("Partial product on thread %d block %d: %lf\r\n", threadIdx.x, blockIdx.x, partial[threadIdx.x]);
    }

    // reduce the values in the buffer to have a single value in the zero element of
    // each buffer.  Use the "good" reduction described in the histogram slides
    // Remember to synchronize appropriately.
    // bad way:
    // if (threadIdx.x == 0)
    // {
    //     double sum = 0;
    //     for (int i = 0; i < THREADS_PER_BLOCK; i++)
    //     {
    //         sum += partial[i];
    //     }
    //     partial[0] = sum;
    // }
    // better reduction:
    for (unsigned int i = blockDim.x / 2; i > 0; i /= 2)
    {
        __syncthreads();
        if (threadIdx.x < i)
        {
            partial[threadIdx.x] += partial[threadIdx.x+i];
        }
    }

    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("Partial sums on thread %d block %d: %lf\r\n", threadIdx.x, blockIdx.x, partial[0]);
    }

    // write the partial reduction for each block stored in element zero of the shared
    // buffer, i.e., the value produced by the reduction above, into the proper
    // location for the block in d_c.
    d_c[blockIdx.x] = partial[0];
}

double hdotProduct(double *h_c, double *h_a, double *h_b, int lengthBytes, int lengthElements,
                   int outputSize, int numBlocks, int threadsBlock)
{
    double *d_a, *d_b, *d_c;
    double sum = 0;
    int valsPerThread = 5;

    // Allocate memory on the device for the d_a, d_b and d_c arrays. Note that the
    // lengths of each are in bytes, not doubles.
    //
    // Copy the h_a and h_b arrays to the d_a and d_b arrays on the gpu.  There is no
    // need to copy d_c values as d_c only holds return values.
    hipMalloc((void **)&d_a, lengthBytes);
    hipMalloc((void **)&d_b, lengthBytes);
    hipMalloc((void **)&d_c, outputSize);

    hipMemcpy(d_a, h_a, lengthBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, lengthBytes, hipMemcpyHostToDevice);

    // launch the kernel. Have four warps of 32 threads (128 threads) for each block.
    // If you use print statements make sure to have a cudaDeviceSynchronize();
    // statement after the launch.
    dotProduct<<<numBlocks, threadsBlock>>>(d_c, d_a, d_b, lengthElements, valsPerThread);

    // copy the d_c array from the device into the h_c array.
    // free d_a, d_b and d_c.
    hipMemcpy(h_c, d_c, outputSize, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // sum the values now in h_c to get the final reduction value, and return that from
    // the function.
    for (int i = 0; i < numBlocks; i++)
    {
        sum += h_c[i];
    }

    return sum;
}

int main(int argc, char **args)
{

    // compute necessary values for the problem, such as number threads per block, etc..
    int lengthElements = VECTOR_LENGTH;
    int lengthBytes = lengthElements * (sizeof(double));
    int outputSize = TOTAL_BLOCKS * (sizeof(double));
    double host_dot = 0;
    double device_dot = 0;

    // declare and allocate h_a, h_b and h_c on the host.
    double *h_a, *h_b, *h_c;
    h_a = (double *)malloc(lengthBytes);
    h_b = (double *)malloc(lengthBytes);
    h_c = (double *)malloc(outputSize);

    // initialize h_a and h_b. I initialized one with i, the position in the array
    // being initialized, and the other with 1.
    for (int i = 0; i < VECTOR_LENGTH; i++)
    {
        h_a[i] = 1.0;
        h_b[i] = 1.0;
    }

    // compute and print the sequential solution
    for (int i = 0; i < lengthElements; i++)
    {
        host_dot += h_a[i] * h_b[i];
    }

    printf("host dotProduct: %lf\r\n", host_dot);

    // call hdotProduct, print the value of c returned (which should equal the sequential
    // value printed above, and free h_a, h_b and h_c.
    device_dot = hdotProduct(h_c, h_a, h_b, lengthBytes, lengthElements, outputSize, TOTAL_BLOCKS, THREADS_PER_BLOCK);
    hipDeviceSynchronize();

    printf("device dotProduct: %lf\r\n", device_dot);

    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
